
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

namespace solution {

    __global__ void convolution2D(const float* __restrict__ input, float* __restrict__ output, int numRows, int numCols, const float* __restrict__ kernel) {
       
        __shared__ float tile[3][3];

       
        if (threadIdx.x < 3 && threadIdx.y < 3) {
            tile[threadIdx.y][threadIdx.x] = kernel[threadIdx.y * 3 + threadIdx.x];
        }
        __syncthreads();

      
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < numRows && col < numCols) {
            float pixel_value = 0.0f;
            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    int r = row + i - 1;
                    int c = col + j - 1;
                    if (r >= 0 && r < numRows && c >= 0 && c < numCols) {
                        pixel_value += input[r * numCols + c] * tile[i][j];
                    }
                }
            }
            output[row * numCols + col] = pixel_value;
        }
    }

    std::string compute(const std::string &bitmap_path, const float kernel[3][3], const std::int32_t num_rows, const std::int32_t num_cols) {
        std::string sol_path = std::filesystem::temp_directory_path() / "student_sol.bmp";
        
        int bitmap_fd = open(bitmap_path.c_str(), O_RDONLY);
        float* img = reinterpret_cast<float*>(mmap(nullptr, sizeof(float) * num_rows * num_cols, PROT_READ, MAP_PRIVATE, bitmap_fd, 0));

      
        float *d_input, *d_output, *d_kernel;
        hipMalloc(&d_input, sizeof(float) * num_rows * num_cols);
        hipMalloc(&d_output, sizeof(float) * num_rows * num_cols);
        hipMalloc(&d_kernel, sizeof(float) * 3 * 3);

  
        hipMemcpy(d_input, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, kernel, sizeof(float) * 3 * 3, hipMemcpyHostToDevice);

      
        dim3 blockDim(32, 32);
        dim3 gridDim((num_cols + blockDim.x - 1) / blockDim.x, (num_rows + blockDim.y - 1) / blockDim.y);

        
        convolution2D<<<gridDim, blockDim>>>(d_input, d_output, num_rows, num_cols, d_kernel);
        
        
        float *output = new float[num_rows * num_cols];
        hipMemcpy(output, d_output, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost);

       
        std::ofstream sol_fs(sol_path, std::ios::binary);
        sol_fs.write(reinterpret_cast<const char*>(output), sizeof(float) * num_rows * num_cols);
        sol_fs.close();

        
        return sol_path;
    }
};
